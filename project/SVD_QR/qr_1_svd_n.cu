/* Compute the SVD of a matrix */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#define ROWS 100000
#define COLS 300
//#define ROWS 31568
//#define COLS 51

#define FILENAME "data.txt"
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define TOTAL_ITERATIONS 5

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void fill(float *p, int n) {
    // This will be replaced by retrieving the data...
    for (int i = 0; i < n; i++) {
        p[i] = (float) (2.0*drand48() + 1.0);
    }
}

void print_matrix(int m, int n, float *A, int lda, const char *name) {
    printf("================A===============================\n");
    for(int row = 0; row < m; row++) {
        for(int col = 0; col < n; col++) {
            float Areg = A[row + col*lda];
            printf("%f\t", Areg);
        }
        printf("\n");
    }
    printf("================end of A===============================\n");
}

void readMatrixFromFile(float *p, int lda){
    FILE *myFile;
    char *filename=FILENAME;
    myFile = fopen(filename, "r");
    if (myFile == NULL)
    {
        printf("Error Reading File\n");
        exit (0);
    }

    char *line=NULL;
    char *word=NULL;
    float attr;
    size_t len = 0;
    ssize_t read;
    int row,col;

    //fill the matrix
    row=0;
    while (((read = getline(&line, &len, myFile)) != -1) && row<ROWS) {
        col=0;
        do{
            word=strsep(&line,",");
            attr = atof(word);
            p[row + col*lda]=attr;
            col++;
        }while(line!=NULL && word!=NULL && col<COLS);
        row++;        
    }  
}

void computeRfromA(float *A, float *R, int lda, int ldr){
    for(int i=0; i< ldr; i++){
        for(int j=0; j< ldr; j++){
            if( i <= j)
                R[i+j*ldr] = A[i+j*lda];
            else
                R[i+j*ldr] = 0.0f;
        }
    }
}

int main(int argc, char *argv[])
{
    
    printf("with my modifications \n");
    hipsolverHandle_t cudenseH = NULL;

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 

    /*====================== used for timing purposes   ======================  */

    hipEvent_t start, stop;
    float time_SVD=0.0f;
    float time_temp;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    /* ======================   Matrix definitions ====================== */

    const int rows = ROWS;
    const int cols = COLS;
    const int mat_A_size = rows*cols;
    const int mat_Q_size = rows*cols;
    const int mat_R_size = cols*cols;
    const int mat_TAU_size = MIN(rows,cols);
    const int mat_U_size = cols*cols;
    const int mat_S_size = cols;
    const int mat_VT_size = cols*cols;

    size_t size_A = mat_A_size*sizeof(float);
    size_t size_TAU = mat_TAU_size*sizeof(float);
    size_t size_R = mat_R_size*sizeof(float);
    size_t size_S = mat_S_size*sizeof(float);
    size_t size_U = mat_U_size*sizeof(float);
    size_t size_VT = mat_VT_size*sizeof(float);

    float *h_A = (float*)malloc(size_A);
    float *h_R = (float*)malloc(size_R);

    /*  copy back only S and Vt after n_iterations  */
    float *h_S = (float*)malloc(size_S);
    float *h_U = (float*)malloc(size_U);
    float *h_VT = (float*)malloc(size_VT);

    /* ====================== matrixes in device ====================== */
    float *d_A = NULL;
    float *d_R = NULL;
    float *d_TAU_QR = NULL;
    float *d_work_QR = NULL;
    float *d_U = NULL;
    float *d_S = NULL;
    float *d_VT = NULL;
    float *d_work_SVD = NULL;
    float *r_work_SVD = NULL;

    /* other variables required for computations */
    int *devInfo_QR = NULL; 
    int *devInfo_SVD = NULL; 
    int info_gpu_QR = 0;
    int info_gpu_SVD =0;
    int lwork_size_QR = 0;
    int lda_QR = rows;
    int lda_SVD=cols;
    int lwork_size_SVD = 0;

    
    

    //fill(h_A, mat_A_size);
    readMatrixFromFile(h_A, rows);

    /*
    printf("A\n");
    print_matrix(rows, cols, h_A, rows, "A");
    printf("\n\n\n");
    */

    /*  ====================== initialise CUDA handle =========================== */

    cusolver_status = hipsolverDnCreate(&cudenseH);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);


    /* ====================Create data structures for device ==================== */

    /* for QR */
    cudaStat1 = hipMalloc((void**)&d_A, size_A);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMalloc((void**)&d_R, size_R);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMalloc ((void**)&d_TAU_QR, size_TAU);   
    assert(hipSuccess == cudaStat1); 

    cudaStat1 = hipMalloc((void**)&devInfo_QR, sizeof(int));
    assert(hipSuccess == cudaStat1);


    /* for SVD */
    cudaStat1 = hipMalloc((void**)&d_U, size_U);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMalloc((void**)&d_S, size_S);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMalloc((void**)&d_VT, size_VT);
    assert(hipSuccess == cudaStat1);

    
    cudaStat1 = hipMalloc((void**)&devInfo_SVD, sizeof(int));
    assert(hipSuccess == cudaStat1);


    /* ======================copy data to device ======================*/

    cudaStat1 = hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);


    /*  ====================== Compute QR ======================    */


    /* calculate the sizes needed for pre-allocated buffer Lwork  */
    cusolver_status = hipsolverDnSgeqrf_bufferSize(cudenseH, rows, cols, d_A, lda_QR, &lwork_size_QR );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work_QR, lwork_size_QR);
    assert(hipSuccess == cudaStat1);

    hipsolverDnSgeqrf( cudenseH, rows, cols, d_A, lda_QR, d_TAU_QR, d_work_QR, lwork_size_QR, devInfo_QR );

    /* check if QR is good or not  */
    cudaStat1 = hipMemcpy(&info_gpu_QR, devInfo_QR, sizeof(int), hipMemcpyDeviceToHost); 
    assert(hipSuccess == cudaStat1);
    assert(0 == info_gpu_QR);

    /* copy A to host */
    cudaStat1 = hipMemcpy(h_A, d_A, size_A, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    computeRfromA(h_A, h_R, lda_QR, lda_SVD);

    /*  ====================== End of QR ======================    */


    /*  ====================== Compute SVD ======================    */

    int rows_SVD=cols;
    int cols_SVD=cols;
    int ldu_SVD=cols;
    int ldvt_SVD=cols;
    char jobu = 'A'; // We do not want/need U
    char jobvt = 'A'; // We want all the vectors of VT

    /*compute buffer size for SVD */
    cusolver_status = hipsolverDnSgesvd_bufferSize(cudenseH, rows_SVD, cols_SVD, &lwork_size_SVD );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    /* for timing purposes only */
    int total_iterations = TOTAL_ITERATIONS;
    hipEvent_t events_start[total_iterations];
    hipEvent_t events_stop[total_iterations];
    for(int i=0;i<total_iterations;i++){
        hipEventCreate(&events_start[i]);
        hipEventCreate(&events_stop[i]);
    }
    int n_iterations;
    for(n_iterations =0; n_iterations < total_iterations; n_iterations ++){

        cudaStat1 = hipMemcpy(d_R, h_R, size_R, hipMemcpyHostToDevice);
        assert(hipSuccess == cudaStat1); 

        cudaStat1 = hipMalloc((void**)&d_work_SVD, lwork_size_SVD);
        assert(hipSuccess == cudaStat1);  

        cudaStat1 = hipMalloc((void**)&r_work_SVD, lwork_size_SVD);
        assert(hipSuccess == cudaStat1);       

        hipEventRecord(events_start[n_iterations], 0);
        cusolver_status = hipsolverDnSgesvd (cudenseH, jobu, jobvt, rows_SVD, cols_SVD, d_R, lda_SVD, d_S, d_U, ldu_SVD, d_VT, ldvt_SVD, d_work_SVD, lwork_size_SVD, r_work_SVD, devInfo_SVD);

        hipEventRecord(events_stop[n_iterations], 0);
        hipEventSynchronize(events_stop[n_iterations]);

        /* check if SVD is good or not  */
        cudaStat1 = hipMemcpy(&info_gpu_SVD, devInfo_SVD, sizeof(int), hipMemcpyDeviceToHost); 
        assert(hipSuccess == cudaStat1);
        assert(0 == info_gpu_SVD);

        /* calculate time for this iteration */
        hipEventElapsedTime(&time_temp, events_start[n_iterations], events_stop[n_iterations]);
        /*
        printf ("Time for the kernel: %f ms\n", time_temp);
        printf ("\n\n\n");
        */

        time_SVD+=time_temp;

        /* free cudaResources */
        if (d_work_SVD ) hipFree(d_work_SVD);

    }

    float Mflop_rate;
    printf ("Time for the kernel: %f ms\n", time_SVD);
    Mflop_rate = 1e-6 * 4 * cols * cols * cols * n_iterations / time_SVD;
    printf ("n_iterations = %d\n",n_iterations);
    printf ("Mflop/s: %f\n", Mflop_rate);

    printf("hipsolverDnSgesvd status :\t");
    switch(cusolver_status)
      {
        case HIPSOLVER_STATUS_SUCCESS:
          printf("success\n");
          break;
        case HIPSOLVER_STATUS_NOT_INITIALIZED :
          printf("Library cuSolver not initialized correctly\n");
          break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
          printf("Invalid parameters passed\n");
          break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
          printf("Internal operation failed\n");
          break;
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
          printf("Execution failed\n");
          break;
      }


    /* ================END of SVD Computation======================= */

    /*  check if SVD is good or not  */
    cudaStat1 =hipMemcpy(&info_gpu_SVD,devInfo_SVD,sizeof(int),hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    printf("after SVD: info_gpu = %d\n", info_gpu_SVD);
    assert(0 == info_gpu_SVD); 
    
    ///*  copy the solutions back to the host */
    cudaStat1 = hipMemcpy(h_R, d_R, size_R, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(h_U, d_U, size_U, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1); 
    cudaStat1 = hipMemcpy(h_S, d_S, size_S, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(h_VT, d_VT, size_VT, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    /* 
    printf("U\n");
    print_matrix(rows, rows, h_U, rows, "A");
    printf("\n\n\n");

    printf("S\n");
    print_matrix(rows, cols, h_S, rows, "S");
    printf("\n\n\n");

    printf("VT\n");
    print_matrix(cols, cols, h_VT, cols, "VT");
    printf("\n\n\n");

     
    printf("A\n");
    print_matrix(rows, rows, h_A, rows, "A");
    printf("\n\n\n");
    */

    /* free resources */
    if (d_A ) hipFree(d_A);
    if (d_TAU_QR ) hipFree(d_TAU_QR);
    if (d_R ) hipFree(d_R);
    if (d_S ) hipFree(d_S);
    if (d_U ) hipFree(d_U);
    if (d_VT ) hipFree(d_VT);

    if(h_A) free(h_A);
    if(h_R) free(h_R);
    if(h_S) free(h_S);
    if(h_U) free(h_U);
    if(h_VT) free(h_VT);

    if (cudenseH) hipsolverDnDestroy(cudenseH);

    

    hipDeviceReset();
}
